/* Reference code implementing numerical integration.
 *
 * Build and execute as follows:
        make clean && make
        ./trap a b n

 * Author: Naga Kandasamy
 * Date modified: February 28, 2025

 * Student name(s): Ricky Chen, Keith Trungcao
 * Date modified: 3/2/25
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

/* Include the kernel code */
#include "trap_kernel.cu"

double compute_on_device(float, float, int, float);
extern "C" double compute_gold(float, float, int, float);

int main(int argc, char **argv)
{
    if (argc < 4) {
        fprintf(stderr, "Usage: %s a b n\n", argv[0]);
        fprintf(stderr, "a: Start limit. \n");
        fprintf(stderr, "b: end limit\n");
        fprintf(stderr, "n: number of trapezoids\n");
        exit(EXIT_FAILURE);
    }

    int a = atoi(argv[1]); /* Left limit */
    int b = atoi(argv[2]); /* Right limit */
    int n = atoi(argv[3]); /* Number of trapezoids */

    float h = (b-a)/(float)n; // Height of each trapezoid
    printf("Number of trapezoids = %d\n", n);
    printf("Height of each trapezoid = %f \n", h);

    double reference = compute_gold(a, b, n, h);
    printf("Reference solution computed on the CPU = %f \n", reference);

    /* Write this function to complete the trapezoidal on the GPU. */
    double gpu_result = compute_on_device(a, b, n, h);
    printf("Solution computed on the GPU = %f \n", gpu_result);
}

/* Complete this function to perform the trapezoidal rule on the GPU. */
double compute_on_device(float a, float b, int n, float h)
{
    double *d_result;
    double h_result = 0.0;

    hipMalloc((void**)&d_result, sizeof(double));

    hipMemcpy(d_result, &h_result, sizeof(double), hipMemcpyHostToDevice);

    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    trap_kernel<<<num_blocks, threads_per_block>>>(a, b, h, n, d_result);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_result);

    return h_result * h;
}



