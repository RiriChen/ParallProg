#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void blur_filter_kernel (const float *in, float *out, int size)
{
    int pix = blockIdx.x * blockDim.x + threadIdx.x;
    int row = pix/size;
    int col = pix%size;
    int curr_row, curr_col;
    float blur_value;
    int num_neighbors;
    int i, j;

    /* Apply blur filter to current pixel */
    blur_value = 0.0;
    num_neighbors = 0;
    for (i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
        for (j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
            /* Accumulate values of neighbors while checking for
             * boundary conditions */
            curr_row = row + i;
            curr_col = col + j;
            if ((curr_row > -1) && (curr_row < size) &&\
                    (curr_col > -1) && (curr_col < size)) {
                blur_value += in[curr_row * size + curr_col];
                num_neighbors += 1;
            }
        }
    }

    /* Write averaged blurred value out */
    out[pix] = blur_value/num_neighbors;

    return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
