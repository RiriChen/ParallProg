#include "hip/hip_runtime.h"
/* Host code for the Jacobi method of solving a system of linear equations
 * by iteration.

 * Build as follows: make clean && make

 * Author: Naga Kandasamy
 * Date modified: March 6, 2025
 *
 * Student name(s); Ricky Chen, Keith Trungcao
 * Date modified: 3/13/25
*/

#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "jacobi_iteration.h"

/* Include the kernel code */
#include "jacobi_iteration_kernel.cu"

struct timeval start, stop;

/* Uncomment the line below if you want the code to spit out debug information. */
/* #define DEBUG */

int main(int argc, char **argv)
{
    if (argc > 1) {
        printf("This program accepts no arguments\n");
        exit(EXIT_FAILURE);
    }

    matrix_t  A;                    /* N x N constant matrix */
    matrix_t  B;                    /* N x 1 b matrix */
    matrix_t reference_x;           /* Reference solution */
    matrix_t gpu_naive_solution_x;  /* Solution computed by naive kernel */
    matrix_t gpu_opt_solution_x;    /* Solution computed by optimized kernel */

    /* Initialize the random number generator */
    srand(time(NULL));

    /* Generate diagonally dominant matrix */
    printf("\nGenerating %d x %d system\n", MATRIX_SIZE, MATRIX_SIZE);
    A = create_diagonally_dominant_matrix(MATRIX_SIZE, MATRIX_SIZE);
    if (A.elements == NULL) {
        printf("Error creating matrix\n");
        exit(EXIT_FAILURE);
    }

    /* Create the other vectors */
    B = allocate_matrix_on_host(MATRIX_SIZE, 1, 1);
    reference_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
    gpu_naive_solution_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
    gpu_opt_solution_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);

#ifdef DEBUG
    print_matrix(A);
    print_matrix(B);
    print_matrix(reference_x);
#endif

    /* Compute Jacobi solution on CPU */
    printf("\nPerforming Jacobi iteration on the CPU\n");
    gettimeofday(&start, NULL);
    compute_gold(A, reference_x, B);
    gettimeofday(&stop, NULL);
    fprintf(stderr, "Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));
    display_jacobi_solution(A, reference_x, B); /* Display statistics */

    /* Compute Jacobi solution on device. Solutions are returned
       in gpu_naive_solution_x and gpu_opt_solution_x. */
    printf("\nPerforming Jacobi iteration on device\n");
    compute_on_device(A, gpu_naive_solution_x, gpu_opt_solution_x, B);
    printf("Naive jacobi:\n");

    display_jacobi_solution(A, gpu_naive_solution_x, B); /* Display statistics */

    printf("optimzied jacobi:\n");

    display_jacobi_solution(A, gpu_opt_solution_x, B);

    free(A.elements);
    free(B.elements);
    free(reference_x.elements);
    free(gpu_naive_solution_x.elements);
    free(gpu_opt_solution_x.elements);

    exit(EXIT_SUCCESS);
}


/* FIXME: Complete this function to perform Jacobi calculation on device */
void compute_on_device(const matrix_t A, matrix_t gpu_naive_sol_x,
                       matrix_t gpu_opt_sol_x, const matrix_t B)
{
    matrix_t A_d = allocate_matrix_on_device(A);
    matrix_t B_d = allocate_matrix_on_device(B);

    printf("\nRunning Naive Implementation\n");
    gettimeofday(&start, NULL);
    matrix_t x_naive_d = allocate_matrix_on_device(gpu_naive_sol_x);
    matrix_t new_x_naive_d = allocate_matrix_on_device(gpu_naive_sol_x);

    double *ssd_naive_d;
    hipMalloc((void **)&ssd_naive_d, sizeof(double) * ((A.num_rows + THREAD_BLOCK_SIZE - 1) / THREAD_BLOCK_SIZE));

    copy_matrix_to_device(A_d, A);
    copy_matrix_to_device(B_d, B);
    copy_matrix_to_device(x_naive_d, gpu_naive_sol_x);

    dim3 threads(THREAD_BLOCK_SIZE, 1, 1);
    dim3 grid((A.num_rows + THREAD_BLOCK_SIZE - 1) / THREAD_BLOCK_SIZE, 1, 1);

    int done_naive = 0;
    double ssd_naive = 0.0;

    while (!done_naive) {
        jacobi_iteration_kernel_naive<<<grid, threads, THREAD_BLOCK_SIZE * sizeof(double)>>>(
            A_d.elements, B_d.elements, x_naive_d.elements, new_x_naive_d.elements, ssd_naive_d, A.num_rows, A.num_columns);

        double *ssd_naive_h = (double *)malloc(grid.x * sizeof(double));
        hipMemcpy(ssd_naive_h, ssd_naive_d, grid.x * sizeof(double), hipMemcpyDeviceToHost);

        ssd_naive = 0.0;
        for (int i = 0; i < grid.x; i++) {
            ssd_naive += ssd_naive_h[i];
        }
        free(ssd_naive_h);

        //printf("ssd_naive: %lf\n", sqrt(ssd_naive));
        //threshold 1-e5
        if (sqrt(ssd_naive) <= THRESHOLD) {
            //printf("done naiive\n");
            done_naive = 1;
        }

        matrix_t tmp = x_naive_d;
        x_naive_d = new_x_naive_d;
        new_x_naive_d = tmp;
    }

    gettimeofday(&stop, NULL);
    fprintf(stderr, "Naive execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));
    check_CUDA_error("Error in kernel");

    copy_matrix_from_device(gpu_naive_sol_x, x_naive_d);


    //matrix_t A_d = allocate_matrix_on_device(A);
    //fprintf(stderr, "\nConverting A to B = A^T to get elements ordered in column-major fashion\n");

    printf("\nRunning coalesced Implementation\n");
    // printf("This is A_T\n");
    // print_matrix(A_T);
    gettimeofday(&start, NULL);

    matrix_t A_T  = allocate_matrix_on_host(A.num_rows, A.num_columns, 0);
    int i, j;
    for (i = 0; i < A_T.num_rows; i++) {
       for (j = 0; j < A_T.num_columns; j++) {
        A_T.elements[i * A_T.num_columns + j] = A.elements[j * A_T.num_columns + i];
       }
    }

    matrix_t A_T_gpu = allocate_matrix_on_device(A_T);

    matrix_t x_opt_d = allocate_matrix_on_device(gpu_opt_sol_x);
    matrix_t new_x_opt_d = allocate_matrix_on_device(gpu_opt_sol_x);

    double *ssd_opt_d;
    hipMalloc((void **)&ssd_opt_d, sizeof(double) * ((A.num_rows + THREAD_BLOCK_SIZE - 1) / THREAD_BLOCK_SIZE));

    copy_matrix_to_device(A_T_gpu, A_T);
    copy_matrix_to_device(B_d, B);
    copy_matrix_to_device(x_opt_d, gpu_opt_sol_x);


    int done_opt = 0;
    double ssd_opt = 0.0;


    while (!done_opt) {

        jacobi_iteration_kernel_optimized_coalesced<<<grid, threads, THREAD_BLOCK_SIZE * sizeof(double)>>>(
            A_T_gpu.elements, B_d.elements, x_opt_d.elements, new_x_opt_d.elements, ssd_opt_d, A_T_gpu.num_rows, A_T_gpu.num_columns);

        double *ssd_opt_h = (double *)malloc(grid.x * sizeof(double));
        hipMemcpy(ssd_opt_h, ssd_opt_d, grid.x * sizeof(double), hipMemcpyDeviceToHost);

        ssd_opt = 0.0;
        for (int i = 0; i < grid.x; i++) {
            ssd_opt += ssd_opt_h[i];
        }
        free(ssd_opt_h);

        if (sqrt(ssd_opt) <= THRESHOLD) {
            done_opt = 1;
        }

        matrix_t tmp = x_opt_d;
        x_opt_d = new_x_opt_d;
        new_x_opt_d = tmp;
    }

    gettimeofday(&stop, NULL);
    fprintf(stderr, "Coalesced execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

    check_CUDA_error("Error in kernel");

    // /* Copy result from the device */
    copy_matrix_from_device(gpu_opt_sol_x, x_opt_d);

    /* Free memory on host */
    free((void *)A_T.elements);
    hipFree(A_T_gpu.elements);
    hipFree(B_d.elements);
    hipFree(x_opt_d.elements);
    hipFree(new_x_opt_d.elements);
    hipFree(ssd_opt_d);
    hipFree(A_d.elements);
    //hipFree(B_d.elements);
    hipFree(x_naive_d.elements);
    hipFree(new_x_naive_d.elements);
    hipFree(ssd_naive_d);
    return;
}


/* Allocate matrix on the device of same size as M */
matrix_t allocate_matrix_on_device(const matrix_t M)
{
    matrix_t Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void **)&Mdevice.elements, size);
    return Mdevice;
}

/* Allocate a matrix of dimensions height * width.
   If init == 0, initialize to all zeroes.
   If init == 1, perform random initialization.
*/
matrix_t allocate_matrix_on_host(int num_rows, int num_columns, int init)
{
    matrix_t M;
    M.num_columns = num_columns;
    M.num_rows = num_rows;
    int size = M.num_rows * M.num_columns;

    M.elements = (float *)malloc(size * sizeof(float));
    for (unsigned int i = 0; i < size; i++) {
        if (init == 0)
            M.elements[i] = 0;
        else
            M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
    }

    return M;
}

/* Copy matrix to device */
void copy_matrix_to_device(matrix_t Mdevice, const matrix_t Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
    return;
}

/* Copy matrix from device to host */
void copy_matrix_from_device(matrix_t Mhost, const matrix_t Mdevice)
{
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
    return;
}

/* Prints the matrix out to screen */
void print_matrix(const matrix_t M)
{
    for (unsigned int i = 0; i < M.num_rows; i++) {
        for (unsigned int j = 0; j < M.num_columns; j++) {
            printf("%f ", M.elements[i * M.num_columns + j]);
        }

        printf("\n");
    }

    printf("\n");
    return;
}

/* Returns a floating-point value between [min, max] */
float get_random_number(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
    return (float)floor((double)(min + (max - min + 1) * r));
}

/* Check for errors in kernel execution */
void check_CUDA_error(const char *msg)
{
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err) {
        printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return;
}

/* Create diagonally dominant matrix */
matrix_t create_diagonally_dominant_matrix(unsigned int num_rows, unsigned int num_columns)
{
    matrix_t M;
    M.num_columns = num_columns;
    M.num_rows = num_rows;
    unsigned int size = M.num_rows * M.num_columns;
    M.elements = (float *)malloc(size * sizeof(float));
    if (M.elements == NULL)
        return M;

    /* Create a matrix with random numbers between [-.5 and .5] */
    unsigned int i, j;
    for (i = 0; i < size; i++)
        M.elements[i] = get_random_number (MIN_NUMBER, MAX_NUMBER);

    /* Make diagonal entries large with respect to the entries on each row. */
    for (i = 0; i < num_rows; i++) {
        float row_sum = 0.0;
        for (j = 0; j < num_columns; j++) {
            row_sum += fabs(M.elements[i * M.num_rows + j]);
        }

        M.elements[i * M.num_rows + i] = 0.5 + row_sum;
    }

    return M;
}
